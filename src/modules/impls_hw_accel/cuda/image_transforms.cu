#include "hip/hip_runtime.h"
#include "image_transforms.h"
#include <cstring>
using namespace std;

__global__ void kernel_crop_memcpy(matrix* img_src, matrix* img_dest, uint2 src_offset)
{
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= img_dest->width || y >= img_dest->height)
    {
        return;
    }

    unsigned char* old_pixel = img_src->get(x + src_offset.x, y + src_offset.y);
    unsigned char* new_pixel = img_dest->get(x, y);
    for (size_t i = 0; i < img_src->components_num; i++)
    {
        new_pixel[i] = old_pixel[i];
    }
}

__global__ void kernel_rotate_memcpy(matrix* img_src, matrix* img_dest, unsigned angle)
{
    unsigned int x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x >= img_src->width || y >= img_src->height)
    {
        return;
    }

    unsigned char* old_pixel = img_src->get(x, y); 
    unsigned char* new_pixel = nullptr;  
    
    switch (angle) {
        case 90:
            new_pixel = img_dest->get(img_dest->width - y - 1, x);
            break;
        case 180:
            new_pixel = img_dest->get(img_dest->width - x - 1, img_dest->height - y -1);
            break;
        case 270:
            new_pixel = img_dest->get(y, img_dest->height - x - 1);
            break;
    }

    for (size_t i = 0; i < img_src->components_num; i++)
    {
        new_pixel[i] = old_pixel[i];
    }
}

void crop(matrix& img, unsigned crop_left, unsigned crop_top, unsigned crop_right, unsigned crop_bottom) 
{
    unsigned new_width = img.width - crop_left - crop_right;
    unsigned new_height = img.height - crop_top - crop_bottom;
    unsigned new_interlaced_size = new_width * new_height * img.components_num;

    if (new_width <= 0 || new_height <= 0) 
    {
        return;
    }

    if (crop_left + crop_right > img.width || crop_top + crop_bottom > img.height) 
    {
        return;
    }

    unsigned total_blocksize = 32;
    if (img.size() >= 4480)
    {
        total_blocksize = 128;
    }

    if (img.size() >= 8960)
    {
        total_blocksize = 256;
    }

    if (img.size() >= 17920)
    {
        total_blocksize = 512;
    }

    if (img.size() >= 35840)
    {
        total_blocksize = 1024;
    }

    matrix* d_img;
    hipMalloc(&d_img, sizeof(matrix));
    hipMemcpy(d_img, &img, sizeof(matrix), hipMemcpyHostToDevice);

    unsigned char* d_cropped_arr;
    hipMalloc(&d_cropped_arr, sizeof(unsigned char) * new_interlaced_size);
    img.set_arr_interlaced(d_cropped_arr, new_width, new_height);

    matrix* d_img_cropped;
    hipMalloc(&d_img_cropped, sizeof(matrix));
    hipMemcpy(d_img_cropped, &img, sizeof(matrix), hipMemcpyHostToDevice);

    int blocksize_2d = (int)(total_blocksize/img.components_num);
    int blocksize_1d = (int)sqrt(blocksize_2d);

    int blocksnum_x = (int)(new_width / blocksize_1d + 1);
    int blocksnum_y = (int)(new_height / blocksize_1d + 1);

    dim3 blockSize(blocksize_1d, blocksize_1d, img.components_num);
    dim3 gridSize(blocksnum_x, blocksnum_y);

    kernel_crop_memcpy<<<gridSize, blockSize>>>(d_img, d_img_cropped, uint2 { crop_right, crop_top });

    hipMemcpy(&img, d_img_cropped, sizeof(matrix), hipMemcpyDeviceToHost);
    hipFree(d_img);
    hipFree(d_img_cropped);
}

void rotate(matrix& img, unsigned angle) 
{
    angle = angle % 360;  
    if (angle == 0) return; 

    unsigned new_width = (angle == 90 || angle == 270) ? img.height : img.width;
    unsigned new_height = (angle == 90 || angle == 270) ? img.width : img.height;

    unsigned total_blocksize = 32;
    if (img.size() >= 4480)
    {
        total_blocksize = 128;
    }

    if (img.size() >= 8960)
    {
        total_blocksize = 256;
    }

    if (img.size() >= 17920)
    {
        total_blocksize = 512;
    }

    if (img.size() >= 35840)
    {
        total_blocksize = 1024;
    }

    int blocksize_2d = (int)(total_blocksize/img.components_num);
    int blocksize_1d = (int)sqrt(blocksize_2d);

    int blocksnum_x = (int)(img.width / blocksize_1d + 1);
    int blocksnum_y = (int)(img.height / blocksize_1d + 1);

    dim3 blockSize(blocksize_1d, blocksize_1d, img.components_num);
    dim3 gridSize(blocksnum_x, blocksnum_y);

    matrix* d_img;
    hipMalloc(&d_img, sizeof(matrix));
    hipMemcpy(d_img, &img, sizeof(matrix), hipMemcpyHostToDevice);

    unsigned char* d_rotated_arr;
    hipMalloc(&d_rotated_arr, sizeof(unsigned char) * img.size_interlaced());
    img.set_arr_interlaced(d_rotated_arr, new_width, new_height);

    matrix* d_img_rotated;
    hipMalloc(&d_img_rotated, sizeof(matrix));
    hipMemcpy(d_img_rotated, &img, sizeof(matrix), hipMemcpyHostToDevice);

    kernel_rotate_memcpy<<<gridSize, blockSize>>>(d_img, d_img_rotated, angle);

    hipMemcpy(&img, d_img_rotated, sizeof(matrix), hipMemcpyDeviceToHost);
    hipFree(d_img);
    hipFree(d_img_rotated);
}
